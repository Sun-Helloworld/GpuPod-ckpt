
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdint.h>

// using device-side mallocs that persist across kernel invocations
// based off of http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#allocation-persisting-kernel-launches
#define NUM_BLOCKS 1
__device__ int* tenptr[NUM_BLOCKS];

__global__ void alloc_and_set_ten()
{
    // Only the first thread in the block does the allocation
    // since we want only one allocation per block.
    if (threadIdx.x == 0) {
        tenptr[blockIdx.x] = (int*)malloc(sizeof(int));
        printf("tenptr[blockIdx.x] = %p\n", tenptr[blockIdx.x]);
        *tenptr[blockIdx.x] = 10;  // set the value
        printf("*tenptr[blockIdx.x] = %d\n", *tenptr[blockIdx.x]);
    }
    __syncthreads();
}
__global__ void copydata(int *dest) 
{
    if (threadIdx.x == 0) {
        // *(dest + blockIdx.x) = *tenptr[blockIdx.x];
        *dest  = 5;
    }
    
}


__global__ void add(int a, int b, int *c)
{   
    printf("add func called \n");
    printf("*tenptr[blockIdx.x] = %d\n", *tenptr[0]);
	*c = a+b+*tenptr[blockIdx.x];
    
}

__global__ void free_ten()
{
    // Free from the leader thread in each thread block
    if (threadIdx.x == 0)
        free(tenptr[blockIdx.x]);
}

int main(int argc, char **argv)
{
	// test
    int a = 2, b = 3, c, *e;
    // uint64_t d;
	int *cuda_c = NULL;
    hipHostMalloc(&e, sizeof(int), hipHostMallocDefault);
    *e=1;

    // 设置设备变量cuda_c
	hipMalloc(&cuda_c, sizeof(int));
    // 设置device变量tenptr为10
    alloc_and_set_ten<<<NUM_BLOCKS,1>>>();
    // cudaDeviceSynchronize();
    printf("sleep start now\n");
    sleep(4);
    printf("sleep end\n");
    
// 这块东西在干什么？
    // void * ptr = (void *)0x7fffe4bd4697;
    // cudaMemcpy(&d, ptr, 8, cudaMemcpyDeviceToHost);
	// printf("%zx\n", d);
    
    add<<<NUM_BLOCKS,1>>>(a, b, cuda_c);

    // printf("cuda_c:%d\n",*cuda_c);   加上这一句出现段错误

    //这里cuda_c 变成15
    
	hipMemcpy(&c, cuda_c, sizeof(int), hipMemcpyDeviceToHost);
	// cudaMemcpy(&e, tenptr[0], sizeof(int), cudaMemcpyDeviceToHost);
    // copydata<<<1,1>>>(e);

    // printf("e:%d\n",*e);
	printf("%d + %d + 10 = %d\n", a, b, c);

	hipFree(cuda_c);
    free_ten<<<NUM_BLOCKS,1>>>();
    printf("end\n");
	exit(EXIT_SUCCESS);
}
