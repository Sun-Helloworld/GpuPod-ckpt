
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void add_2_1(int a, int b, int *c)
{
  *c = a + b;
}

int main(int argc, char **argv)
{
  // test
  int a, b, *c = NULL;
  // int a, b, *c = NULL,*d = NULL;

  // Test for read faults
  hipMallocManaged(&c, sizeof(int));
  // cudaMallocManaged(&d, sizeof(int));
  a = 2;
  b = 3;
  // *d = 5;
  sleep(5); // Allow time to checkpoint
  add_2_1<<<1,1>>>(a, b, c);
  hipDeviceSynchronize();

  printf("%d + %d = %d\n", a, b, *c);
  // printf("d = %d\n", *d);

  exit(EXIT_SUCCESS);
}
