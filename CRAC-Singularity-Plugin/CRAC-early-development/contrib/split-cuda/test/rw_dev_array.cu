
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

// reading and writing a device variable (with array)
__device__ int incr_idx = 0;
__device__ int incr_arr [] = {10, 11};

__global__ void add(int a, int b, int *c)
{
	*c = a+b+incr_arr[incr_idx++];
	printf("c = %d at %p\n", *c, c);
}

__global__ void addd(int a, int b, int *c)
{	
	printf("👿cuda_d addd start \n ");

	*c = a+b+5;
	printf("👿d = %d at %p\n", *c, c);
	printf("👿cuda_d addd end \n ");
}
__global__ void aprint(int a, int b, int *c)
{
	*c = a+b+5;
	printf("d = %d at %p\n", *c, c);
}

int main(int argc, char **argv)
{
	// test
	int a = 2, b = 3, c=2 ,d =2;
	int *cuda_c = NULL;
	int *cuda_d = NULL;
	printf("c val = %d\n", c);


	printf("cuda_c before malloc= %p\n", cuda_c);

	hipMalloc(&cuda_c, sizeof(int));
	hipMalloc(&cuda_d, sizeof(int));


	printf("cuda_c = %p\n", cuda_c);
	printf("cuda_d = %p\n", cuda_d);
	// add<<<1,1>>>(a, b, cuda_c);
	add<<<1,1>>>(a, b, cuda_c);
	hipDeviceSynchronize();

	printf("cuda_c add = %p\n", cuda_c);
	printf("c val = %d\n", c);





	printf("before checkpoint\n");
    sleep(4);
	printf("after checkpoint\n");
	


	printf("cuda_c add = %p\n", cuda_c);
	printf("c val = %d\n", c);

	// 这个有问题
	hipMemcpy(&c, cuda_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("cuda_d add = %p\n", cuda_d);


	addd<<<1,1>>>(a, b, cuda_d);
	// printf("cuda_d add = %d\n", *(int*)cuda_d);
	hipDeviceSynchronize();
	hipMemcpy(&d, cuda_d, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	printf("d val = %d\n", d);
	
	

	hipFree(cuda_c);
	hipFree(cuda_d);

	printf("%d + %d + 11= %d\n", a, b, c);

	exit(EXIT_SUCCESS);
}
